#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define X 5
#define Y 5
#define PADDING_SIZE_X 1
//my gpu has 1664 cuda cores
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

//kernel called from the CPU to the GPU to apply the padding
__global__ void applyPadding(int *matrix, int *paddedMatrix, int width, int height, int paddingX) {
    //Set the values of the index of the blocks and threads to set them on each index
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    //If the values of the col or row are out of the size of the matrix
    if (col < width && row  < height) {
        //operation to add the padding every time it passes of how many numbers are in the row
        paddedMatrix[row * (width + paddingX) + col + paddingX] = matrix[row * width + col];
    }
}

//kernel called from the CPU to the GPU to add the columns
__global__ void sumColumns(int *matrix, int *result, int width, int height) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    //function to add the numbers of the same column
    if (col < width) {
        int sum = 0;
        for (int row = 0; row < height; ++row) {
            sum += matrix[(row) * width + col];
        }
        result[col] = sum;
    }
}

int main() {
    //variables that are going to be used in the cpu to print values
    int matrix[X][Y];
    int paddedMatrix[X + PADDING_SIZE_X][Y];
    int result[Y];

    // Insert random values to matrix
    for (int i = 0; i < X; ++i) {
        for (int j = 0; j < Y; ++j) {
            //We add one so that we can see the padding as 0's and nothing else
            matrix[i][j] = (rand() % 9)+1;
            //for testing purposes to compare to the image in the book
            //matrix[i][j] = j+1;
        }
    }
    // Print original matrix
    printf("Original:\n");
    for (int i = 0; i < X; ++i) {
        for (int j = 0; j < Y; ++j) {
            printf("%d\t", matrix[i][j]);
        }
        printf("\n");
    }

    //GPU variables
    int *d_matrix, *d_paddedMatrix, *d_padMatrix, *d_resultSum;

    //allocates memory of variables of the GPU for the first kernel
    hipMalloc((void **)&d_matrix, X * Y * sizeof(int));
    hipMalloc((void **)&d_paddedMatrix, (X + PADDING_SIZE_X) * Y * sizeof(int));

    //For the second kernel
    hipMalloc((void **)&d_padMatrix, (X + PADDING_SIZE_X) * Y * sizeof(int));
    hipMalloc((void **)&d_resultSum, Y * sizeof(int));

    // Copy the matrix to the variable that is going to be sent to the GPU "d_matrix" with size of X*Y
    hipMemcpy(d_matrix, matrix, X * Y * sizeof(int), hipMemcpyHostToDevice);

    // Define the dimention of block and grid
    dim3 gridDim((Y + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (X + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, 1);
    dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);

    //Call the kernel, sending it the d_matrix and asking for the result back in d_paddedmatrix
    applyPadding<<<gridDim, blockDim>>>(d_matrix, d_paddedMatrix, Y, X, PADDING_SIZE_X);

    //Copy result of the d_paddedMatrix to the variable thats on the CPU so it can print it
    hipMemcpy(paddedMatrix, d_paddedMatrix, (X + PADDING_SIZE_X) * Y * sizeof(int), hipMemcpyDeviceToHost);

    // Print matrix with padding
    printf("\nWith Padding:\n");
    //you have to add the padding size to the for
    for (int i = 0; i < X + PADDING_SIZE_X; ++i) {
        for (int j = 0; j < Y ; ++j) {
            printf("%d\t", paddedMatrix[i][j]);
        }
        printf("\n");
    }

    //Copies the values on the padded matrix we just made and passes it to a variable used for the GPU
    hipMemcpy(d_padMatrix, paddedMatrix, (X + PADDING_SIZE_X) * Y * sizeof(int), hipMemcpyHostToDevice);

    //Calls Kernel for the sum of the values on the columns and places it on d_resultSum
    sumColumns<<<gridDim, blockDim>>>(d_padMatrix, d_resultSum, Y, X + PADDING_SIZE_X);

    // Copies the result it got from adding the columns to the result variable
    hipMemcpy(result, d_resultSum, Y * sizeof(int), hipMemcpyDeviceToHost);

    // Print Sum's
    printf("\nColumn Sums:\n");
    for (int j = 0; j < Y; ++j) {
        printf("%d\t", result[j]);
    }
    printf("\n");

    //Free memory from all the variables of the GPU
    hipFree(d_matrix);
    hipFree(d_paddedMatrix);
    hipFree(d_padMatrix);
    hipFree(d_resultSum);

    return 0;
}